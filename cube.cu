#include "hip/hip_runtime.h"
%%cu
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#include <stdio.h>



__global__ void cube(float *d_out,float *d_in)
{

    int id=threadIdx.x;
    printf("Thread id  : %d\n",id);
    float f=d_in[id];
    d_out[id]=f*f*f;
}

int main()
{
    const int AS=96;
    const int AB=AS*sizeof(float);

    float h_in[AS];

    for(int i=0;i<AS;i++)
    h_in[i]=float(i);

    float h_out[AS];
    float * d_in;
    float * d_out;
    hipMalloc((void **)&d_in,AB);
    hipMalloc((void **)&d_out,AB);
    hipMemcpy(d_in,h_in,AB,hipMemcpyHostToDevice);
    //square<<<1,AS>>>(d_out,d_in);
    cube<<<dim3(1,1,1),dim3(AS,1,1)>>>(d_out,d_in);
    hipMemcpy(h_out,d_out,AB,hipMemcpyDeviceToHost);
    for(int i=0;i<AS;i++)
    printf("%f,",h_out[i]);
    hipFree(d_in);
    hipFree(d_out);
    return 0;

}