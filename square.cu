#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <time.h>
#include <stdio.h>

// #define N 1<<20
// #define THREADS 1024
// #define BLOCKS 1024

__global__ void square(float *d_out,float *d_in)
{

    int id=threadIdx.x;
    float f=d_in[id];
    d_out=f*f;
}


int main()
{
    int AS=64;
    int AB=AS*sizeof(float);

    float h_in[AS];

    for(int i=0;i<AS;i++)
    h_in[i]=float(i);

    float h_out[AS];
    float *d_in;
    float *d_out;
    hipMalloc((void **)&d_in,AB);
    hipMalloc((void **)&d_out,AB);
    hipMemcpy(d_in,h_in,AB,hipMemcpyHostToDevice);
    square<<<1,AS>>>(d_out,d_in);
    hipMemcpy(h_out,d_out,AB,hipMemcpyDeviceToHost);
    for(int i=0;i<AS;i++)
    cout<<h_out[i]<<",";
    hipFree(d_in);
    hipFree(d_out);
    return 0;

}